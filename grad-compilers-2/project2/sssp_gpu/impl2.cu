#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <algorithm>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "graph.h"
#include "parse_graph.hpp"
#include "limits.h"

/****** START UTIL METHODS ******/
bool edgeSrcComparator(edge a, edge b){ return (a.src < b.src); }
bool edgeDestComparator(edge a, edge b){ return (a.dest < b.dest); }

void swap(void **a, void **b){
    void *tmp = *a;
    *a = *b;
    *b = tmp;
}

int readCudaInt(int *i){
    int tmp;
    hipMemcpy(&tmp, i, sizeof(int), hipMemcpyDeviceToHost);
    
    return tmp;
}

void printEdges(std::vector<edge> edges){
    for(edge e : edges){
        printf("src: %d, dst: %d, weight: %d\n", e.src, e.dest, e.weight);
    }
}

int getNumVertices(std::vector<edge> edges){
    int max = -1;
    for(edge e : edges){
        int tmp = std::max(e.src, e.dest);
        max = std::max(max, tmp);
    }

    return max + 1;
}

void writeAnswer(int *output, int len){
    FILE *fp = fopen("output.txt", "w");
    for(int i = 0; i < len; i++){
        fprintf(fp, "%d:\t%d\n", i, output[i]);
    }
    fclose(fp);
}

__global__ void cudaInitIntArray(int *a, int len, int val){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (len % totalWarps == 0) ? len / totalWarps : len / totalWarps + 1;
    int beg = load * warpId;
    int end = (len < beg + load) ? len : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        a[i] = val;
    }
}

/****** END UTIL METHODS ******/

int INF = INT_MAX;
int ZERO = 0;

__global__ void setToMin(int *distPrev, int *distCur, int distLen){ 
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (distLen % totalWarps == 0) ? distLen / totalWarps : distLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (distLen < beg + load) ? distLen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        atomicMin(&distCur[i], distPrev[i]);
        atomicMin(&distPrev[i], distCur[i]);
    }
}

__global__ void filterEdges(edge *edges, int edgesLen, edge *toProcessEdges, int *x, int *updated){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (edgesLen % totalWarps == 0) ? edgesLen / totalWarps : edgesLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (edgesLen < beg + load) ? edgesLen : beg + load;
    beg = beg + laneId;
    int curOffset = x[warpId];

    for(int i = beg; i < end; i += 32){
        int src = edges[i].src;
        int mask = __ballot(updated[src]);
        int localId = __popc(mask<<(32-laneId));
        if(updated[src]){
            memcpy(&toProcessEdges[localId + curOffset], &edges[i], sizeof(edge));
        }
        curOffset += __popc(mask);
    }
}

__global__ void getExcPrefixSum(int *x, int *toProcessLen, int totalWarps){
    int prevSum = 0;
    for(int i = 0; i < totalWarps; i++){
        int tmp = x[i];
        x[i] = prevSum;
        prevSum += tmp;
    }

    // update length of toProcess edges
    *toProcessLen = prevSum;
}

__global__ void getNumToProcess(edge *edges, int edgesLen, int *x, int *updated){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (edgesLen % totalWarps == 0) ? edgesLen / totalWarps : edgesLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (edgesLen < beg + load) ? edgesLen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        int src = edges[i].src;
        int mask = __ballot(updated[src]);

        if(laneId == 0){
            x[warpId] += __popc(mask);
        }
    }
}

__global__ void processEdges2(edge *edges, int *edgesLen, int *distPrev, int *distCur, int *hasUpdated, int *updated){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (*edgesLen % totalWarps == 0) ? *edgesLen / totalWarps : *edgesLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (*edgesLen < beg + load) ? *edgesLen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        int src = edges[i].src;
        int dest = edges[i].dest;
        int weight = edges[i].weight;

        // avoid overflow when adding weight to src
        if(distPrev[src] == INT_MAX)
            continue;
        
        int dist = distPrev[src] + weight;
        if(dist < distPrev[dest]){
            atomicMin(&distCur[dest], dist);
            *hasUpdated = 1;
            updated[dest] = 1;
        }
    }
}

__global__ void processEdgesIncore2(edge *edges, int *edgesLen, int *dist, int *hasUpdated, int *updated){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (*edgesLen % totalWarps == 0) ? *edgesLen / totalWarps : *edgesLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (*edgesLen < beg + load) ? *edgesLen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        int src = edges[i].src;
        int dest = edges[i].dest;
        int weight = edges[i].weight;

        // avoid overflow when adding weight to src
        if(dist[src] == INT_MAX)
            continue;
        
        int tmpDist = dist[src] + weight;
        if(tmpDist < dist[dest]){
            atomicMin(&dist[dest], tmpDist);
            *hasUpdated = 1;
            updated[dest] = 1;
        }
    }
}

void bmfOutcore2(std::vector<edge> edges, int blockSize, int blockNum){
    int numVertices = getNumVertices(edges);
    int numEdges = edges.size();
    int totalThreads = blockSize * blockNum;
    int warpNum = (totalThreads % 32 == 0) ? totalThreads / 32 : totalThreads / 32 + 1;
    int distanceVectorSize = sizeof(int) * numVertices;

    edge *edges_d;
    hipMalloc((void**)&edges_d, numEdges * sizeof(edge));
    hipMemcpy(edges_d, edges.data(), edges.size()*sizeof(edge), hipMemcpyHostToDevice);

    int *distCur;
    hipMalloc((void**)&distCur, distanceVectorSize);
    hipMemcpy((void*)distCur, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    cudaInitIntArray<<<blockNum, blockSize>>>(&distCur[1], numVertices - 1, INF);

    int *distPrev;
    hipMalloc((void**)&distPrev, distanceVectorSize);
    hipMemcpy((void*)distPrev, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    cudaInitIntArray<<<blockNum, blockSize>>>(&distPrev[1], numVertices - 1, INF);

    int *hasUpdated;
    hipMalloc((void**)&hasUpdated, sizeof(int));
    hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);

    int *x; 
    hipMalloc((void**)&x, sizeof(int) * warpNum);
    cudaInitIntArray<<<blockNum, blockSize>>>(x, warpNum, ZERO);

    edge *toProcessEdges;
    hipMalloc((void**)&toProcessEdges, numEdges * sizeof(edge));
    hipMemcpy(toProcessEdges, edges.data(), edges.size()*sizeof(edge), hipMemcpyHostToDevice);

    int *toProcessEdgesLen;
    hipMalloc((void**)&toProcessEdgesLen, sizeof(int));
    hipMemcpy(toProcessEdgesLen, &numEdges, sizeof(int), hipMemcpyHostToDevice);

    int *updated;
    hipMalloc((void**)&updated, sizeof(int) * numVertices);
    cudaInitIntArray<<<blockNum, blockSize>>>(updated, numVertices, ZERO);

    // start BMF
    while(true){
        processEdges2<<<blockNum, blockSize>>>(toProcessEdges, toProcessEdgesLen, distPrev, distCur, hasUpdated, updated);
        getNumToProcess<<<blockNum, blockSize>>>(edges_d, numEdges, x, updated);
        getExcPrefixSum<<<1, 1>>>(x, toProcessEdgesLen, warpNum);
        filterEdges<<<blockNum, blockSize>>>(edges_d, numEdges, toProcessEdges, x, updated);
        setToMin<<<blockNum, blockSize>>>(distPrev, distCur, numVertices);

        if(!readCudaInt(hasUpdated))
            break;

        // reset for next iteration
        cudaInitIntArray<<<blockNum, blockSize>>>(x, warpNum, ZERO);
        cudaInitIntArray<<<blockNum, blockSize>>>(updated, numVertices, ZERO);
        hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);
        swap((void**)&distCur, (void**)&distPrev);
    }

    // write answer to file
    int *output = (int*) malloc(distanceVectorSize);
    hipMemcpy((void*)output, distCur, distanceVectorSize, hipMemcpyDeviceToHost);
    writeAnswer(output, numVertices);

    // free up
    hipFree(edges_d);
    hipFree(distPrev);
    hipFree(distCur);
    hipFree(hasUpdated);
    hipFree(x);
    hipFree(toProcessEdges);
    hipFree(toProcessEdgesLen);
    hipFree(updated);
    free(output);
}

void bmfIncore2(std::vector<edge> edges, int blockSize, int blockNum){
    int numVertices = getNumVertices(edges);
    int numEdges = edges.size();
    int totalThreads = blockSize * blockNum;
    int warpNum = (totalThreads % 32 == 0) ? totalThreads / 32 : totalThreads / 32 + 1;
    int distanceVectorSize = sizeof(int) * numVertices;

    edge *edges_d;
    hipMalloc((void**)&edges_d, numEdges * sizeof(edge));
    hipMemcpy(edges_d, edges.data(), edges.size()*sizeof(edge), hipMemcpyHostToDevice);

    int *dist;
    hipMalloc((void**)&dist, distanceVectorSize);
    hipMemcpy((void*)dist, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    cudaInitIntArray<<<blockNum, blockSize>>>(&dist[1], numVertices - 1, INF);

    int *hasUpdated;
    hipMalloc((void**)&hasUpdated, sizeof(int));
    hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);

    int *x; 
    hipMalloc((void**)&x, sizeof(int) * warpNum);
    cudaInitIntArray<<<blockNum, blockSize>>>(x, warpNum, ZERO);

    edge *toProcessEdges;
    hipMalloc((void**)&toProcessEdges, numEdges * sizeof(edge));
    hipMemcpy(toProcessEdges, edges.data(), edges.size()*sizeof(edge), hipMemcpyHostToDevice);

    int *toProcessEdgesLen;
    hipMalloc((void**)&toProcessEdgesLen, sizeof(int));
    hipMemcpy(toProcessEdgesLen, &numEdges, sizeof(int), hipMemcpyHostToDevice);

    int *updated;
    hipMalloc((void**)&updated, sizeof(int) * numVertices);
    cudaInitIntArray<<<blockNum, blockSize>>>(updated, numVertices, ZERO);

    // start BMF
    while(true){
        processEdgesIncore2<<<blockNum, blockSize>>>(toProcessEdges, toProcessEdgesLen, dist, hasUpdated, updated);
        getNumToProcess<<<blockNum, blockSize>>>(edges_d, numEdges, x, updated);
        getExcPrefixSum<<<1, 1>>>(x, toProcessEdgesLen, warpNum);
        filterEdges<<<blockNum, blockSize>>>(edges_d, numEdges, toProcessEdges, x, updated);

        if(!readCudaInt(hasUpdated))
            break;

        // reset for next iteration
        cudaInitIntArray<<<blockNum, blockSize>>>(x, warpNum, ZERO);
        cudaInitIntArray<<<blockNum, blockSize>>>(updated, numVertices, ZERO);
        hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    }

    // write answer to file
    int *output = (int*) malloc(distanceVectorSize);
    hipMemcpy((void*)output, dist, distanceVectorSize, hipMemcpyDeviceToHost);
    writeAnswer(output, numVertices);

    // free up
    hipFree(edges_d);
    hipFree(dist);
    hipFree(hasUpdated);
    hipFree(x);
    hipFree(toProcessEdges);
    hipFree(toProcessEdgesLen);
    hipFree(updated);
    free(output);
}

void neighborHandler(std::vector<edge> * edgesPtr, int blockSize, int blockNum, int outcore){
    setTime();

    if(!((blockSize * blockNum <= 2048) && (blockNum <= 64))){
        puts("ERROR: blockNum must be <= 64 and total threads must be <= 2048\n");
        return;
    }
    std::vector<edge> edges = *edgesPtr;
    std::sort(edges.begin(), edges.end(), edgeSrcComparator);
    outcore ? bmfOutcore2(edges, blockSize, blockNum) : bmfIncore2(edges, blockSize, blockNum);

    hipDeviceProp_t props; hipGetDeviceProperties(&props, 0);
    printf("The total computation kernel time on GPU %s is %f milli-seconds\n", props.name, getTime());
}
