#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <algorithm>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "parse_graph.hpp"
#include "graph.h"
#include "limits.h"

__global__ void processEdges(edge *edges, int edgesLen, int *distPrev, int *distCur, int *hasUpdated){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (edgesLen % totalWarps == 0) ? edgesLen / totalWarps : edgesLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (edgesLen < beg + load) ? edgesLen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        int src = edges[i].src;
        int dest = edges[i].dest;
        int weight = edges[i].weight;

        // avoid overflow when adding weight to src
        if(distPrev[src] == INT_MAX)
            continue;
        
        int dist = distPrev[src] + weight;
        if(dist < distPrev[dest]){
            atomicMin(&distCur[dest], dist);
            *hasUpdated = 1;
        }
    }
}

void bmfOutcore(std::vector<edge> edges, int blockSize, int blockNum){
    int numVertices = getNumVertices(edges);
    int numEdges = edges.size();
    int distanceVectorSize = sizeof(int) * numVertices;

    edge *edges_d;
    hipMalloc((void**)&edges_d, numEdges * sizeof(edge));
    hipMemcpy(edges_d, edges.data(), edges.size()*sizeof(edge), hipMemcpyHostToDevice);

    int *distCur;
    hipMalloc((void**)&distCur, distanceVectorSize);
    hipMemcpy((void*)distCur, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    cudaInitIntArray<<<blockNum, blockSize>>>(&distCur[1], numVertices - 1, INF);

    int *distPrev;
    hipMalloc((void**)&distPrev, distanceVectorSize);
    hipMemcpy((void*)distPrev, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    cudaInitIntArray<<<blockNum, blockSize>>>(&distPrev[1], numVertices - 1, INF);

    int *hasUpdated;
    hipMalloc((void**)&hasUpdated, sizeof(int));
    hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);

    // start BMF
    while(true){
        processEdges<<<blockNum, blockSize>>>(edges_d, numEdges, distPrev, distCur, hasUpdated);
        setToMin<<<blockNum, blockSize>>>(distPrev, distCur, numVertices);

        if(!readCudaInt(hasUpdated))
            break;

        // reset for next iteration
        hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);
        swap((void**)&distCur, (void**)&distPrev);
    }

    // write answer to file
    int *output = (int*) malloc(distanceVectorSize);
    hipMemcpy((void*)output, distCur, distanceVectorSize, hipMemcpyDeviceToHost);
    writeAnswer(output, numVertices);

    // free up
    hipFree(edges_d);
    hipFree(distPrev);
    hipFree(distCur);
    hipFree(hasUpdated);
    free(output);
}

__global__ void processEdgesIncore(edge *edges, int edgesLen, int *dist, int *hasUpdated){
    int totalThreads = gridDim.x * blockDim.x;
    int totalWarps = (totalThreads % 32 == 0) ?  totalThreads / 32 : totalThreads / 32 + 1;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadId / 32;
    int laneId = threadId % 32;
    int load = (edgesLen % totalWarps == 0) ? edgesLen / totalWarps : edgesLen / totalWarps + 1;
    int beg = load * warpId;
    int end = (edgesLen < beg + load) ? edgesLen : beg + load;
    beg = beg + laneId;

    for(int i = beg; i < end; i += 32){
        int src = edges[i].src;
        int dest = edges[i].dest;
        int weight = edges[i].weight;

        // avoid overflow when adding weight to src
        if(dist[src] == INT_MAX)
            continue;
        
        int tmpDist = dist[src] + weight;
        if(tmpDist < dist[dest]){
            atomicMin(&dist[dest], tmpDist);
            *hasUpdated = 1;
        }
    }
}

void bmfIncore(std::vector<edge> edges, int blockSize, int blockNum){
    int numVertices = getNumVertices(edges);
    int numEdges = edges.size();
    int distanceVectorSize = sizeof(int) * numVertices;

    edge *edges_d;
    hipMalloc((void**)&edges_d, numEdges * sizeof(edge));
    hipMemcpy(edges_d, edges.data(), edges.size()*sizeof(edge), hipMemcpyHostToDevice);

    int *dist;
    hipMalloc((void**)&dist, distanceVectorSize);
    hipMemcpy((void*)dist, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    cudaInitIntArray<<<blockNum, blockSize>>>(&dist[1], numVertices - 1, INF);

    int *hasUpdated;
    hipMalloc((void**)&hasUpdated, sizeof(int));
    hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);

    // start BMF
    while(true){
        processEdgesIncore<<<blockNum, blockSize>>>(edges_d, numEdges, dist, hasUpdated);

        if(!readCudaInt(hasUpdated))
            break;

        // reset for next iteration
        hipMemcpy((void*)hasUpdated, &ZERO, sizeof(int), hipMemcpyHostToDevice);
    }

    // write answer to file
    int *output = (int*) malloc(distanceVectorSize);
    hipMemcpy((void*)output, dist, distanceVectorSize, hipMemcpyDeviceToHost);
    writeAnswer(output, numVertices);

    // free up
    hipFree(edges_d);
    hipFree(dist);
    hipFree(hasUpdated);
    free(output);
}

void puller(std::vector<edge> *edgesPtr, int blockSize, int blockNum, int outcore){
    setTime();

    std::vector<edge> edges = *edgesPtr;
    std::sort(edges.begin(), edges.end(), edgeSrcComparator);
    outcore ? bmfOutcore(edges, blockSize, blockNum) : bmfIncore(edges, blockSize, blockNum);
    
    hipDeviceProp_t props; hipGetDeviceProperties(&props, 0);
    printf("The total computation kernel time on GPU %s is %f milli-seconds\n", props.name, getTime());
}
